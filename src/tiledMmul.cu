#include "hip/hip_runtime.h"
// This program computes matrix multiplication using shared memory tiling
// By: Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

#include <nvToolsExt.h>

using std::cout;
using std::generate;
using std::vector;

// Pull out matrix and shared memory tile size 
const int N = 1 << 10;
const int SHMEM_SIZE = 1 << 10;

__global__ void tiledMatrixMul(const int *a, const int *b, int *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  c[row * N + col] = tmp;
}

// Check result on the CPU
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}

int main() {
  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  nvtxRangePush("allocate host memory for three matrices");
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);
  nvtxRangePop();

  // Initialize matrices
  nvtxRangePush("initialize two source matrices with random numbers");
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });
  nvtxRangePop();

  // Allocate device memory
  nvtxRangePush("allocate device memory for three matrices");
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  nvtxRangePop();

  // Copy data to the device
  nvtxRangePush("copy matrices from host to device memory");
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);
  nvtxRangePop();

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  std::cout << "Launch Kernel: " << THREADS << " threads per block, " << BLOCKS << " blocks in the grid" << std::endl;
  nvtxRangePush("start kernel");
  tiledMatrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
  nvtxRangePop();

  // Copy back to the host
  nvtxRangePush("copy matrix from device to host memory");
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);
  nvtxRangePop();

  // Check result
  nvtxRangePush("verify result");
  verify_result(h_a, h_b, h_c);
  nvtxRangePop();

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  nvtxRangePush("free device memory");
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  nvtxRangePop();

  return 0;
}
